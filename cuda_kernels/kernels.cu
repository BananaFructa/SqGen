#include "hip/hip_runtime.h"
#include "kernels.cuh"

#include <math.h>

#include <stdio.h>

#define MAX(a,b) (a > b ? a : b)

hipStream_t *currentStream = NULL;

__global__ void addTensor_kernel(Tensor_DEVICE tTarget, Tensor_DEVICE tSource1, Tensor_DEVICE tSource2, size_t elemSize1, size_t elemSize2, int operand) {
	size_t i = threadIdx.x + blockIdx.x * blockDim.x;
	size_t maxSize = max(elemSize1, elemSize2);
	if (i < maxSize) {
		tTarget[i] = tSource1[i % elemSize1] + tSource2[i % elemSize2] + operand * tTarget[i];
	}
}

__global__ void hadamardTensor_kernel(Tensor_DEVICE tTarget, Tensor_DEVICE tSource1, Tensor_DEVICE tSource2, size_t elemSize1, size_t elemSize2, int operand) {
	size_t i = threadIdx.x + blockIdx.x * blockDim.x;
	size_t maxSize = max(elemSize1, elemSize2);
	if (i < maxSize) {
		tTarget[i] = tSource1[i % elemSize1] * tSource2[i % elemSize2] + operand * tTarget[i];
	}
}

// documentation just for this since is more important
/*
* @param tTarget = The tensor pool in which the results will be stored
* @param tSource1 = First tensor pool in the multiplication operation
* @param tSource2 = Second tensor pool in the multiplication operation
* @param poolSize = The tensor pool size
* @param prodLc = The product between the lines of the first and the columns of the second tensor
* @param l = The number of lines of all the tenors in the first pool
* @param cl = The number of lines/columns of all the tensors from the first/second pool
* @param c = The number of columns of all the tensors in the second pool
* @param single = True if tSource2 is a single tensor and every tensor from tSource1 should be multiplied with it
*/
__global__ void mulTensor_kernel(Tensor_DEVICE tTarget,
								 Tensor_DEVICE tSource1,
	                             Tensor_DEVICE tSource2,
	                             size_t poolSize1,
								 size_t poolSize2,
								 size_t prodLc,
	                             size_t l,
	                             size_t cl,
	                             size_t c,
								 int operand
) {
	size_t t = threadIdx.x + blockIdx.x * blockDim.x;
	size_t tensorStep = t % prodLc;
	size_t poolId = t / prodLc;

	if (poolId < max(poolSize1, poolSize2)) {

		size_t line = tensorStep / c;
		size_t column = tensorStep % c;
		TENSOR_TYPE sum = 0;

		for (size_t i = 0; i < cl; i++) {
			sum += tSource1[(poolId % poolSize1) * prodLc + line + i * l] * tSource2[(poolId % poolSize2) * prodLc + i + column * cl];
		}

		size_t targetId = poolId * prodLc + line + column * l;
		tTarget[targetId] = sum + operand * tTarget[targetId];

	}
}

__global__ void funcPassReLU_kernel(Tensor_DEVICE t, Func f, size_t size) {
	size_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < size) {
		bool p = t[i] > 0;
		t[i] = t[i] * p + t[i] * 0.1f * !p;
	}
}

__global__ void funcPassSigmoid_kernel(Tensor_DEVICE t, Func f, size_t size) {
	size_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < size) {
		t[i] = 1.0f / (1.0f + expf(-(float)t[i]));
	}
}

__global__ void funcPassTanh_kernel(Tensor_DEVICE t, Func f, size_t size) {
	size_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < size) {
		t[i] = tanhf((float)t[i]);
	}
}

__global__ void funcPassExp_kernel(Tensor_DEVICE t, Func f, size_t size) {
	size_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < size) {
		t[i] = expf((float)t[i]);
	}
}

__global__ void normalizeTensor_kernel(Tensor_DEVICE t, Tensor_DEVICE sum, size_t poolSize, size_t elemSize) {
	size_t i = threadIdx.x + blockIdx.x * blockDim.x;
	size_t tensorStep = i % elemSize;
	size_t poolId = i / elemSize;
	size_t tensorNumber = poolId * elemSize;
	if (poolId < poolSize) {
		t[tensorNumber + tensorStep] /= sum[tensorNumber];
	}
}

__global__ void sumTensor_kernel(Tensor_DEVICE t, Tensor_DEVICE sum, size_t poolSize, size_t elemSize) {
	size_t i = threadIdx.x + blockIdx.x * blockDim.x;
	size_t tensorStep = i % elemSize;
	size_t poolId = i / elemSize;
	size_t tensorNumber = (i / elemSize) * elemSize;
	if (poolId < poolSize) {
		atomicAdd(&sum[tensorNumber], t[tensorNumber + tensorStep]);
	}
}

__global__ void curandInit_kernel(hiprandState_t* state, size_t size, unsigned long seed) {
	size_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < size) {
		hiprand_init(seed, i, 0, &state[i]);
	}
}

__global__ void randomizeTensorUniform_kernel(hiprandState_t* state, Tensor_DEVICE t, size_t size, float low, float absoluteDifference) {
	size_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < size) {
		t[i] = hiprand_uniform(&state[i]) * absoluteDifference + low;
	}
}

__global__ void rndOffsetTensorUniform_kernel(hiprandState_t* state, Tensor_DEVICE t, size_t size, float prob, float low, float absoluteDifference) {
	size_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < size && hiprand_uniform(&state[i]) <= prob) {
		t[i] += hiprand_uniform(&state[i]) * absoluteDifference + low;
	}
}

Tensor_DEVICE allocateTensor(size_t size) {
	Tensor_DEVICE tensor;
	hipMalloc(&tensor, size * sizeof(TENSOR_TYPE));
	return tensor;
}

void bindTensor(hipStream_t *stream) {
	currentStream = stream;
}

void freeTensor(Tensor_DEVICE t) {
	hipFree(t);
}

void copyTensorFromDevice(Tensor_HOST tHost,Tensor_DEVICE t, size_t size) {
	hipMemcpy(tHost, t, size * sizeof(TENSOR_TYPE), hipMemcpyDeviceToHost);
}

void copyTensorFromHost(Tensor_HOST tHost, Tensor_DEVICE t, size_t size) {
	hipMemcpy(t, tHost, size * sizeof(TENSOR_TYPE), hipMemcpyHostToDevice);
}

void CudaKernels::addTensor(Tensor_DEVICE tTarget, Tensor_DEVICE tSource1, Tensor_DEVICE tSource2,size_t elemSize1,size_t elemSize2,int operand) {

	dim3 threadSize(256);
	dim3 blockSize((MAX(elemSize1,elemSize2) + threadSize.x - 1) / threadSize.x);

	addTensor_kernel <<< blockSize, threadSize, 0, (currentStream ? *currentStream : 0) >>> (tTarget, tSource1, tSource2, elemSize1, elemSize2,operand);
}

void CudaKernels::hadamardTensor(Tensor_DEVICE tTarget, Tensor_DEVICE tSource1, Tensor_DEVICE tSource2, size_t elemSize1, size_t elemSize2, int operand) {

	dim3 threadSize(256);
	dim3 blockSize((MAX(elemSize1, elemSize2) + threadSize.x - 1) / threadSize.x);

	hadamardTensor_kernel << < blockSize, threadSize, 0, (currentStream ? *currentStream : 0) >> > (tTarget, tSource1, tSource2, elemSize1, elemSize2,operand);
}

void CudaKernels::funcPass(Tensor_DEVICE t, Func f, size_t size) {

	dim3 threadSize(256);
	dim3 blockSize((size + threadSize.x - 1) / threadSize.x);

	switch (f) {
		case KERNEL_ReLU:
			funcPassReLU_kernel << < blockSize, threadSize, 0, (currentStream ? *currentStream : 0) >> > (t, f, size);
			break;
		case KERNEL_SIGMOID:
			funcPassSigmoid_kernel << < blockSize, threadSize, 0, (currentStream ? *currentStream : 0) >> > (t, f, size);
			break;
		case KERNEL_TANH:
			funcPassTanh_kernel << < blockSize, threadSize, 0, (currentStream ? *currentStream : 0) >> > (t, f, size);
			break;
		case KERNEL_EXP:
			funcPassExp_kernel << < blockSize, threadSize, 0, (currentStream ? *currentStream : 0) >> > (t, f, size);
			break;
	}
}

void CudaKernels::normalizeTensor(Tensor_DEVICE t, Tensor_DEVICE sum, size_t poolSize, size_t elemSize) {

	dim3 threadSize(256);
	dim3 blockSize((poolSize * elemSize + threadSize.x - 1) / threadSize.x);

	normalizeTensor_kernel <<< blockSize,threadSize, 0, (currentStream ? *currentStream : 0) >>> (t,sum,poolSize,elemSize);
}

void CudaKernels::sumTensor(Tensor_DEVICE t, Tensor_DEVICE sum, size_t poolSize, size_t elemSize) {
	dim3 threadSize(256);
	dim3 blockSize((poolSize * elemSize + threadSize.x - 1) / threadSize.x);

	sumTensor_kernel <<< blockSize, threadSize, 0, (currentStream ? *currentStream : 0) >>> (t, sum, poolSize, elemSize);
}

void CudaKernels::mulTensor2D(Tensor_DEVICE tTarget,Tensor_DEVICE tSource1, Tensor_DEVICE tSource2,size_t poolSize1,size_t poolSize2, size_t l, size_t cl , size_t c,int operand) {
	size_t processCountPerTensor = l * c;
	
	dim3 threadSize(256);
	dim3 blockSize((processCountPerTensor * MAX(poolSize1,poolSize2) + threadSize.x - 1) / threadSize.x);
	
	mulTensor_kernel <<< blockSize,threadSize,0,(currentStream ? *currentStream : 0) >>> (tTarget, tSource1, tSource2, poolSize1,poolSize2,processCountPerTensor, l, cl, c,operand);
}

void CudaKernels::curandStateAlloc(hiprandState_t* state, size_t size, unsigned long seed) {
	
	dim3 threadSize(256);
	dim3 blockSize((size + threadSize.x - 1) / threadSize.x);

	curandInit_kernel << < blockSize, threadSize, 0, (currentStream ? *currentStream : 0) >> > (state, size, seed);
}

void CudaKernels::randomizeTensorUniform(hiprandState_t* state, Tensor_DEVICE t, size_t size, float lowerRange, float higherRange) {
	dim3 threadSize(256);
	dim3 blockSize((size + threadSize.x - 1) / threadSize.x);
	
	randomizeTensorUniform_kernel << < blockSize, threadSize, 0, (currentStream ? *currentStream : 0) >> > (state, t, size, lowerRange, fabsf(lowerRange - higherRange));
}

void CudaKernels::rndOffsetTensorUniform(hiprandState_t* state, Tensor_DEVICE t, size_t size, float prob, float lowerRange, float higherRange) {
	dim3 threadSize(256);
	dim3 blockSize((size + threadSize.x - 1) / threadSize.x);

	rndOffsetTensorUniform_kernel << < blockSize, threadSize, 0, (currentStream ? *currentStream : 0) >> > (state, t, size, prob, lowerRange, fabsf(lowerRange - higherRange));
}

void gpuSync() {
	hipDeviceSynchronize();
}

void gpuSyncStream(hipStream_t* stream) {
	hipStreamSynchronize(*stream);
}

void bindStream(hipStream_t* stream) {
	currentStream = stream;
}

void createStream(hipStream_t* stream) {
	hipStreamCreate(stream);
}

void destroyStream(hipStream_t* stream) {
	hipStreamDestroy(*stream);
}
