#include "hip/hip_runtime.h"
#include "kernels.cuh"

#include <math.h>

#include <stdio.h>

#define MAX(a,b) (a > b ? a : b)

hipStream_t *currentStream = NULL;

__global__ void initZeroTensor_kernel(Tensor_DEVICE t, size_t size) {
	size_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < size) {
		t[i] = 0;
	}
}

__global__ void initZeroTensorMapped_kernel(TensorMap_DEVICE m, size_t size, size_t blockSize, size_t allignOffset) {
	size_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < size) {
		m[(i + allignOffset) / blockSize][(i + allignOffset) % blockSize] = 0;
	}
}


__global__ void addTensor_kernel(Tensor_DEVICE tTarget, Tensor_DEVICE tSource1, Tensor_DEVICE tSource2, size_t elemSize1, size_t elemSize2, int operand) {
	size_t i = threadIdx.x + blockIdx.x * blockDim.x;
	size_t maxSize = max(elemSize1, elemSize2);
	if (i < maxSize) {
		tTarget[i] = tSource1[i % elemSize1] + tSource2[i % elemSize2] + operand * tTarget[i];
	}
}

__global__ void addTensorMapped_kernel(TensorMap_DEVICE mapT, TensorMap_DEVICE map1, TensorMap_DEVICE map2, size_t elemSize1, size_t elemSize2, size_t blockSizeT, size_t blockSize1, size_t blockSize2, size_t allignOffsetT, size_t allignOffset1, size_t allignOffset2, int operand) {
	size_t i = threadIdx.x + blockIdx.x * blockDim.x;
	size_t maxSize = max(elemSize1, elemSize2);
	if (i < maxSize) {
		Tensor_DEVICE target = mapT[i / blockSizeT];
		size_t accesPoint1 = i % elemSize1 + allignOffset1;
		size_t accesPoint2 = i % elemSize2 + allignOffset2;
		size_t accesPointT = i + allignOffsetT;
		target[accesPointT % blockSizeT] = map1[accesPoint1 / blockSize1][accesPoint1 % blockSize1] + map2[accesPoint2 / blockSize2][accesPoint2 % blockSize2] + operand * target[accesPointT % blockSizeT];
	}
}

__global__ void hadamardTensor_kernel(Tensor_DEVICE tTarget, Tensor_DEVICE tSource1, Tensor_DEVICE tSource2, size_t elemSize1, size_t elemSize2, int operand) {
	size_t i = threadIdx.x + blockIdx.x * blockDim.x;
	size_t maxSize = max(elemSize1, elemSize2);
	if (i < maxSize) {
		tTarget[i] = tSource1[i % elemSize1] * tSource2[i % elemSize2] + operand * tTarget[i];
	}
}

__global__ void hadamardTensorMapped_kernel(TensorMap_DEVICE mapT, TensorMap_DEVICE map1, TensorMap_DEVICE map2, size_t elemSize1, size_t elemSize2, size_t blockSizeT, size_t blockSize1, size_t blockSize2, size_t allignOffsetT, size_t allignOffset1, size_t allignOffset2, int operand) {
	size_t i = threadIdx.x + blockIdx.x * blockDim.x;
	size_t maxSize = max(elemSize1, elemSize2);
	if (i < maxSize) {
		Tensor_DEVICE target = mapT[i / blockSizeT];
		size_t accesPoint1 = i % elemSize1 + allignOffset1;
		size_t accesPoint2 = i % elemSize2 + allignOffset2;
		size_t accesPointT = i + allignOffsetT;
		target[accesPointT % blockSizeT] = map1[accesPoint1 / blockSize1][accesPoint1 % blockSize1] * map2[accesPoint2 / blockSize2][accesPoint2 % blockSize2] + operand * target[accesPointT % blockSizeT];
	}
}

// documentation just for this since is more important
/*
* @param tTarget = The tensor pool in which the results will be stored
* @param tSource1 = First tensor pool in the multiplication operation
* @param tSource2 = Second tensor pool in the multiplication operation
* @param poolSize = The tensor pool size
* @param prodLc = The product between the lines of the first and the columns of the second tensor
* @param l = The number of lines of all the tenors in the first pool
* @param cl = The number of lines/columns of all the tensors from the first/second pool
* @param c = The number of columns of all the tensors in the second pool
* @param single = True if tSource2 is a single tensor and every tensor from tSource1 should be multiplied with it
*/
__global__ void mulTensor_kernel(Tensor_DEVICE tTarget,
								 Tensor_DEVICE tSource1,
	                             Tensor_DEVICE tSource2,
	                             size_t poolSize1,
								 size_t poolSize2,
								 size_t prodLc,
	                             size_t l,
	                             size_t cl,
	                             size_t c,
								 int operand
) {
	size_t t = threadIdx.x + blockIdx.x * blockDim.x;
	size_t tensorStep = t % prodLc;
	size_t poolId = t / prodLc;

	if (poolId < max(poolSize1, poolSize2)) {

		size_t line = tensorStep / c;
		size_t column = tensorStep % c;
		TENSOR_TYPE sum = 0;

		for (size_t i = 0; i < cl; i++) {
			sum += tSource1[(poolId % poolSize1) * l * cl + line + i * l] * tSource2[(poolId % poolSize2) * cl * c + i + column * cl];
		}

		size_t targetId = poolId * prodLc + line + column * l;
		tTarget[targetId] = sum + operand * tTarget[targetId];

	}
}

__global__ void mulTensorMapped_kernel(TensorMap_DEVICE mapT,
									   TensorMap_DEVICE map1,
									   TensorMap_DEVICE map2,
									   size_t poolSize1,
									   size_t poolSize2,
									   size_t prodLc,
									   size_t l,
									   size_t cl,
									   size_t c,
									   size_t blockSizeT,
									   size_t blockSize1,
									   size_t blockSize2,
									   size_t allignOffsetT,
									   size_t allignOffset1,
									   size_t allignOffset2,
									   int operand
) {
	size_t t = threadIdx.x + blockIdx.x * blockDim.x;
	size_t tensorStep = t % prodLc;
	size_t poolId = t / prodLc;

	if (poolId < max(poolSize1, poolSize2)) {

		size_t line = tensorStep / c;
		size_t column = tensorStep % c;
		TENSOR_TYPE sum = 0;

		size_t accesPoint1, accesPoint2;

		for (size_t i = 0; i < cl; i++) {

			accesPoint1 = (poolId % poolSize1) * l * cl + line + i * l + allignOffset1;
			accesPoint2 = (poolId % poolSize2) * cl * c + i + column * cl + allignOffset2;

			sum += map1[accesPoint1 / blockSize1][accesPoint1 % blockSize1] * map2[accesPoint2 / blockSize2][accesPoint2 % blockSize2];
		}

		size_t targetId = poolId * prodLc + line + column * l + allignOffsetT;
		Tensor_DEVICE val = mapT[targetId / blockSizeT];
		val[targetId % blockSizeT] = sum + operand * val[targetId % blockSizeT];

	}
}

__global__ void funcPassReLU_kernel(Tensor_DEVICE t, size_t size) {
	size_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < size) {
		bool p = t[i] > 0;
		t[i] = t[i] * p + t[i] * 0.1f * !p;
	}
}

__global__ void funcPassMappedReLU_kernel(TensorMap_DEVICE m, size_t blockSize, size_t allignOffset, size_t size) {
	size_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < size) {
		size_t accesPoint = i + allignOffset;
		size_t blockId = accesPoint / blockSize;
		size_t blockIndex = accesPoint % blockSize;
		bool p = m[blockId][blockIndex] > 0;
		m[blockId][blockIndex] = m[blockId][blockIndex] * p + m[blockId][blockIndex] * 0.1f * !p;
	}
}

__global__ void funcPassSigmoid_kernel(Tensor_DEVICE t, size_t size) {
	size_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < size) {
		t[i] = 1.0f / (1.0f + expf(-(float)t[i]));
	}
}

__global__ void funcPassMappedSigmoid_kernel(TensorMap_DEVICE m, size_t blockSize, size_t allignOffset, size_t size) {
	size_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < size) {
		size_t accesPoint = i + allignOffset;
		size_t blockId = accesPoint / blockSize;
		size_t blockIndex = accesPoint % blockSize;
		m[blockId][blockIndex] = 1.0f / (1.0f + expf(-(float)m[blockId][blockIndex]));
	}
}

__global__ void funcPassTanh_kernel(Tensor_DEVICE t, size_t size) {
	size_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < size) {
		t[i] = tanhf((float)t[i]);
	}
}

__global__ void funcPassMappedTanh_kernel(TensorMap_DEVICE m, size_t blockSize, size_t allignOffset, size_t size) {
	size_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < size) {
		size_t accesPoint = i + allignOffset;
		size_t blockId = accesPoint / blockSize;
		size_t blockIndex = accesPoint % blockSize;
		m[blockId][blockIndex] = tanhf((float)m[blockId][blockIndex]);
	}
}

__global__ void funcPassExp_kernel(Tensor_DEVICE t, size_t size) {
	size_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < size) {
		t[i] = expf((float)t[i]);
	}
}

__global__ void funcPassMappedExp_kernel(TensorMap_DEVICE m, size_t blockSize, size_t allignOffset, size_t size) {
	size_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < size) {
		size_t accesPoint = i + allignOffset;
		size_t blockId = accesPoint / blockSize;
		size_t blockIndex = accesPoint % blockSize;
		m[blockId][blockIndex] = expf((float)m[blockId][blockIndex]);
	}
}

__global__ void normalizeTensor_kernel(Tensor_DEVICE t, Tensor_DEVICE sum, size_t poolSize, size_t elemSize) {
	size_t i = threadIdx.x + blockIdx.x * blockDim.x;
	size_t tensorStep = i % elemSize;
	size_t poolId = i / elemSize;
	size_t tensorNumber = poolId * elemSize;
	if (poolId < poolSize) {
		t[tensorNumber + tensorStep] /= sum[tensorNumber];
	}
}

__global__ void normalizeTensorMapped_kernel(TensorMap_DEVICE m, Tensor_DEVICE sum, size_t poolSize, size_t elemSize, size_t blockSize, size_t allignOffset) {
	size_t i = threadIdx.x + blockIdx.x * blockDim.x;
	size_t tensorStep = i % elemSize;
	size_t poolId = i / elemSize;
	size_t tensorNumber = poolId * elemSize;
	if (poolId < poolSize) {
		size_t accesPoint = tensorNumber + tensorStep + allignOffset;
		m[accesPoint / blockSize][accesPoint % blockSize] /= sum[tensorNumber];
	}
}

__global__ void sumTensor_kernel(Tensor_DEVICE t, Tensor_DEVICE sum, size_t poolSize, size_t elemSize) {
	size_t i = threadIdx.x + blockIdx.x * blockDim.x;
	size_t tensorStep = i % elemSize;
	size_t poolId = i / elemSize;
	size_t tensorNumber = (i / elemSize) * elemSize;
	if (poolId < poolSize) {
		atomicAdd(&sum[tensorNumber], t[tensorNumber + tensorStep]);
	}
}

__global__ void sumTensorMapped_kernel(TensorMap_DEVICE m, Tensor_DEVICE sum, size_t poolSize, size_t elemSize, size_t blockSize, size_t allignOffset) {
	size_t i = threadIdx.x + blockIdx.x * blockDim.x;
	size_t tensorStep = i % elemSize;
	size_t poolId = i / elemSize;
	size_t tensorNumber = (i / elemSize) * elemSize;
	if (poolId < poolSize) {
		size_t accesPoint = tensorNumber + tensorStep + allignOffset;
		atomicAdd(&sum[tensorNumber], m[accesPoint / blockSize][accesPoint % blockSize]);
	}
}

__global__ void curandInit_kernel(hiprandState_t* state, size_t size, unsigned long seed) {
	size_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < size) {
		hiprand_init(seed, i, 0, &state[i]);
	}
}

__global__ void randomizeTensorUniform_kernel(hiprandState_t* state, Tensor_DEVICE t, size_t size, float low, float absoluteDifference) {
	size_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < size) {
		t[i] = hiprand_uniform(&state[i]) * absoluteDifference + low;
	}
}

__global__ void rndOffsetTensorUniform_kernel(hiprandState_t* state, Tensor_DEVICE t, size_t size, float prob, float low, float absoluteDifference) {
	size_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < size && hiprand_uniform(&state[i]) <= prob) {
		t[i] += hiprand_uniform(&state[i]) * absoluteDifference + low;
	}
}

AllocRes allocateTensor(size_t size,size_t mapSize) {
	void* tensorData;
	hipMalloc(&tensorData, size * sizeof(TENSOR_TYPE) + mapSize * sizeof(TENSOR_TYPE*));
	return { (Tensor_DEVICE)((TensorMap_DEVICE)tensorData + mapSize),(TensorMap_DEVICE)tensorData };
}

void copyTensorD2D(TensorMap_DEVICE target, TensorMap_DEVICE source, size_t mapSize, size_t tensorSize) {
	hipMemcpy(target, source, tensorSize * sizeof(TENSOR_TYPE) + mapSize * sizeof(TENSOR_TYPE*), hipMemcpyDeviceToDevice);
}

void bindTensor(hipStream_t *stream) {
	currentStream = stream;
}

void freeCudaMem(void* t) {
	hipFree(t);
}

void copyTensorFromDevice(Tensor_HOST tHost,Tensor_DEVICE t, size_t size) {
	hipMemcpy(tHost, t, size * sizeof(TENSOR_TYPE), hipMemcpyDeviceToHost);
}

void copyTensorFromHost(Tensor_HOST tHost, Tensor_DEVICE t, size_t size) {
	hipMemcpy(t, tHost, size * sizeof(TENSOR_TYPE), hipMemcpyHostToDevice);
}

void copyMapFromHost(TensorMap_DEVICE mHost, TensorMap_DEVICE m, size_t size) {
	hipMemcpy(m, mHost, size * sizeof(TENSOR_TYPE*), hipMemcpyHostToDevice);
}

void CudaKernels::addTensor(Tensor_DEVICE tTarget, Tensor_DEVICE tSource1, Tensor_DEVICE tSource2,size_t elemSize1,size_t elemSize2,int operand) {

	dim3 threadSize(256);
	dim3 blockSize((MAX(elemSize1,elemSize2) + threadSize.x - 1) / threadSize.x);

	addTensor_kernel <<< blockSize, threadSize, 0, (currentStream ? *currentStream : 0) >>> (tTarget, tSource1, tSource2, elemSize1, elemSize2,operand);
}

void CudaKernels::addTensorMapped(TensorMap_DEVICE mapT, TensorMap_DEVICE map1, TensorMap_DEVICE map2, size_t elemSize1, size_t elemSize2, size_t blockSizeT, size_t blockSize1, size_t blockSize2, size_t allignOffsetT, size_t allignOffset1, size_t allignOffset2, int operand) {
	dim3 threadSize(256);
	dim3 blockSize((MAX(elemSize1, elemSize2) + threadSize.x - 1) / threadSize.x);

	addTensorMapped_kernel << < blockSize, threadSize, 0, (currentStream ? *currentStream : 0) >> > (mapT,map1,map2,elemSize1,elemSize2,blockSizeT,blockSize1,blockSize2,allignOffsetT,allignOffset1,allignOffset2, operand);
}

void CudaKernels::hadamardTensor(Tensor_DEVICE tTarget, Tensor_DEVICE tSource1, Tensor_DEVICE tSource2, size_t elemSize1, size_t elemSize2, int operand) {

	dim3 threadSize(256);
	dim3 blockSize((MAX(elemSize1, elemSize2) + threadSize.x - 1) / threadSize.x);

	hadamardTensor_kernel << < blockSize, threadSize, 0, (currentStream ? *currentStream : 0) >> > (tTarget, tSource1, tSource2, elemSize1, elemSize2,operand);
}

void CudaKernels::hadamardTensorMapped(TensorMap_DEVICE mapT, TensorMap_DEVICE map1, TensorMap_DEVICE map2, size_t elemSize1, size_t elemSize2, size_t blockSizeT, size_t blockSize1, size_t blockSize2, size_t allignOffsetT, size_t allignOffset1, size_t allignOffset2, int operand) {
	dim3 threadSize(256);
	dim3 blockSize((MAX(elemSize1, elemSize2) + threadSize.x - 1) / threadSize.x);

	hadamardTensorMapped_kernel << < blockSize, threadSize, 0, (currentStream ? *currentStream : 0) >> > (mapT, map1, map2, elemSize1, elemSize2, blockSizeT, blockSize1, blockSize2,allignOffsetT,allignOffset1,allignOffset2, operand);
}

void CudaKernels::funcPass(Tensor_DEVICE t, Func f, size_t size) {

	dim3 threadSize(256);
	dim3 blockSize((size + threadSize.x - 1) / threadSize.x);

	switch (f) {
		case KERNEL_ReLU:
			funcPassReLU_kernel << < blockSize, threadSize, 0, (currentStream ? *currentStream : 0) >> > (t, size);
			break;
		case KERNEL_SIGMOID:
			funcPassSigmoid_kernel << < blockSize, threadSize, 0, (currentStream ? *currentStream : 0) >> > (t, size);
			break;
		case KERNEL_TANH:
			funcPassTanh_kernel << < blockSize, threadSize, 0, (currentStream ? *currentStream : 0) >> > (t, size);
			break;
		case KERNEL_EXP:
			funcPassExp_kernel << < blockSize, threadSize, 0, (currentStream ? *currentStream : 0) >> > (t, size);
			break;
	}
}

void CudaKernels::funcPassMapped(TensorMap_DEVICE m, size_t blockSize_, size_t allignOffset, size_t size, Func f) {
	dim3 threadSize(256);
	dim3 blockSize((size + threadSize.x - 1) / threadSize.x);

	switch (f) {
		case KERNEL_ReLU:
			funcPassMappedReLU_kernel << < blockSize, threadSize, 0, (currentStream ? *currentStream : 0) >> > (m,blockSize_,allignOffset,size);
			break;
		case KERNEL_SIGMOID:
			funcPassMappedSigmoid_kernel << < blockSize, threadSize, 0, (currentStream ? *currentStream : 0) >> > (m, blockSize_, allignOffset, size);
			break;
		case KERNEL_TANH:
			funcPassMappedTanh_kernel << < blockSize, threadSize, 0, (currentStream ? *currentStream : 0) >> > (m, blockSize_, allignOffset, size);
			break;
		case KERNEL_EXP:
			funcPassMappedExp_kernel << < blockSize, threadSize, 0, (currentStream ? *currentStream : 0) >> > (m, blockSize_, allignOffset, size);
			break;
	}
}

void CudaKernels::initZeroTensor(Tensor_DEVICE t, size_t size) {
	dim3 threadSize(256);
	dim3 blockSize((size + threadSize.x - 1) / threadSize.x);

	initZeroTensor_kernel << < blockSize, threadSize, 0, (currentStream ? *currentStream : 0) >> > (t, size);
}

void CudaKernels::initZeroTensorMapped(TensorMap_DEVICE m, size_t size, size_t blockSize_, size_t allignOffset) {
	dim3 threadSize(256);
	dim3 blockSize((size + threadSize.x - 1) / threadSize.x);

	initZeroTensorMapped_kernel << < blockSize, threadSize, 0, (currentStream ? *currentStream : 0) >> > (m,size,blockSize_,allignOffset);
}

void CudaKernels::normalizeTensor(Tensor_DEVICE t, Tensor_DEVICE sum, size_t poolSize, size_t elemSize) {

	dim3 threadSize(256);
	dim3 blockSize((poolSize * elemSize + threadSize.x - 1) / threadSize.x);

	normalizeTensor_kernel <<< blockSize,threadSize, 0, (currentStream ? *currentStream : 0) >>> (t,sum,poolSize,elemSize);
}

void CudaKernels::normalizeTensorMapped(TensorMap_DEVICE m, Tensor_DEVICE sum, size_t poolSize, size_t elemSize, size_t blockSize_, size_t allignOffset) {
	dim3 threadSize(256);
	dim3 blockSize((poolSize * elemSize + threadSize.x - 1) / threadSize.x);

	normalizeTensorMapped_kernel << < blockSize, threadSize, 0, (currentStream ? *currentStream : 0) >> >  (m, sum, poolSize, elemSize, blockSize_, allignOffset);
}

void CudaKernels::sumTensor(Tensor_DEVICE t, Tensor_DEVICE sum, size_t poolSize, size_t elemSize) {
	dim3 threadSize(256);
	dim3 blockSize((poolSize * elemSize + threadSize.x - 1) / threadSize.x);

	sumTensor_kernel <<< blockSize, threadSize, 0, (currentStream ? *currentStream : 0) >>> (t, sum, poolSize, elemSize);
}

void CudaKernels::sumTensorMapped(TensorMap_DEVICE m, Tensor_DEVICE sum, size_t poolSize, size_t elemSize, size_t blockSize_, size_t allignOffset) {
	dim3 threadSize(256);
	dim3 blockSize((poolSize * elemSize + threadSize.x - 1) / threadSize.x);

	sumTensorMapped_kernel <<< blockSize, threadSize, 0, (currentStream ? *currentStream : 0) >>> (m, sum, poolSize, elemSize, blockSize_, allignOffset);
}

void CudaKernels::mulTensor2D(Tensor_DEVICE tTarget,Tensor_DEVICE tSource1, Tensor_DEVICE tSource2,size_t poolSize1,size_t poolSize2, size_t l, size_t cl , size_t c,int operand) {
	size_t processCountPerTensor = l * c;
	
	dim3 threadSize(256);
	dim3 blockSize((processCountPerTensor * MAX(poolSize1,poolSize2) + threadSize.x - 1) / threadSize.x);
	
	mulTensor_kernel <<< blockSize,threadSize,0,(currentStream ? *currentStream : 0) >>> (tTarget, tSource1, tSource2, poolSize1,poolSize2,processCountPerTensor, l, cl, c,operand);
}

void CudaKernels::mulTensorMapped2D(TensorMap_DEVICE tTarget, TensorMap_DEVICE tSource1, TensorMap_DEVICE tSource2, size_t poolSize1, size_t poolSize2, size_t l, size_t cl, size_t c, size_t blockSizeT, size_t blockSize1, size_t blockSize2, size_t allignOffsetT, size_t allignOffset1, size_t allignOffset2, int operand) {
	size_t processCountPerTensor = l * c;

	dim3 threadSize(256);
	dim3 blockSize((processCountPerTensor * MAX(poolSize1, poolSize2) + threadSize.x - 1) / threadSize.x);

	mulTensorMapped_kernel << < blockSize, threadSize, 0, (currentStream ? *currentStream : 0) >> >  (tTarget, tSource1, tSource2, poolSize1, poolSize2, processCountPerTensor, l, cl, c, blockSizeT, blockSize1, blockSize2,allignOffsetT,allignOffset1,allignOffset2, operand);
}

void CudaKernels::curandStateAlloc(hiprandState_t* state, size_t size, unsigned long seed) {
	
	dim3 threadSize(256);
	dim3 blockSize((size + threadSize.x - 1) / threadSize.x);

	curandInit_kernel << < blockSize, threadSize, 0, (currentStream ? *currentStream : 0) >> > (state, size, seed);
}

void CudaKernels::randomizeTensorUniform(hiprandState_t* state, Tensor_DEVICE t, size_t size, float lowerRange, float higherRange) {
	dim3 threadSize(256);
	dim3 blockSize((size + threadSize.x - 1) / threadSize.x);
	
	randomizeTensorUniform_kernel << < blockSize, threadSize, 0, (currentStream ? *currentStream : 0) >> > (state, t, size, lowerRange, fabsf(lowerRange - higherRange));
}

void CudaKernels::rndOffsetTensorUniform(hiprandState_t* state, Tensor_DEVICE t, size_t size, float prob, float lowerRange, float higherRange) {
	dim3 threadSize(256);
	dim3 blockSize((size + threadSize.x - 1) / threadSize.x);

	rndOffsetTensorUniform_kernel << < blockSize, threadSize, 0, (currentStream ? *currentStream : 0) >> > (state, t, size, prob, lowerRange, fabsf(lowerRange - higherRange));
}

void gpuSync() {
	hipDeviceSynchronize();
}

void gpuSyncStream(hipStream_t* stream) {
	hipStreamSynchronize(*stream);
}

void bindStream(hipStream_t* stream) {
	currentStream = stream;
}

void createStream(hipStream_t* stream) {
	hipStreamCreate(stream);
}

void destroyStream(hipStream_t* stream) {
	hipStreamDestroy(*stream);
}
